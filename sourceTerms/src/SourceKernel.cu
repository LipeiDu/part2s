#include "hip/hip_runtime.h"

#include "ParameterReader.cpp"

__host__ __device__ float KernelCartesian(float r0, float r1, float r2, float r3, float ri0, float ri1, float ri2, float ri3, float pi0, float pi1, float pi2, float pi3, float mi, float gi, float gmax, float vmax, float sigma2, float delta_tau, float d_tauInv, float SigInv, float *kernelT){
    
    // ********************************************************
    // if the contraction factor is too large, rescale it
    // ********************************************************
    
    float mInv = 1/mi;
    float u1 = pi1 * mInv;
    float u2 = pi2 * mInv;
    float u3 = pi3 * mInv;
    float u0 = sqrt(1 + u1*u1 + u2*u2 + u3*u3);
    
    float gratio = gmax / gi;
    
    if (gi > gmax)
    {
        float gi2 = gi * gi;
        float vi = sqrt(1-1/gi2);
        float vratio = vmax / vi;
        float ratio = gratio * vratio;
        
        u0 = u0 * gratio;
        u1 = u1 * ratio;
        u2 = u2 * ratio;
        u3 = u3 * ratio;
    }
    
    // ********************************************************
    // calculate the kernel
    // ********************************************************
    
    float kernel = 0.0;
    
    //convert (tau, eta) into (t, z)
    float t = r0 * cosh(r3);
    float z = r0 * sinh(r3);
    
    float d0 = t  - ri0;
    float d1 = r1 - ri1;
    float d2 = r2 - ri2;
    float d3 = z  - ri3;
    
    float dxsqd = d0 * d0 - d1 * d1 - d2 * d2 - d3 * d3;
    float udotx = u0 * d0 - u1 * d1 - u2 * d2 - u3 * d3;
    float dist4d = dxsqd - udotx * udotx;
    
    if (dist4d <= 9 * sigma2 && udotx <= 4 * delta_tau) // if the particle's contribution is important
    {
        float exponent = dist4d * SigInv;
        float exponentiation = exp(exponent);
        
        float uxnorm = udotx * d_tauInv;
        float ch = cosh(uxnorm);
        float chInv = 1/ch;
        
        kernel = gratio * chInv * chInv * exponentiation; // [1], only cosh^2 * exp * gamma term

        if (isnan(kernel)){
            printf("Kernel is nan for this particle, set to 0...\n");
            kernel = 0.0;
        }
        
        *kernelT = gratio * exponentiation;
        
        if (isnan(*kernelT)){
            printf("KernelT is nan for this particle, set to 0...\n");
            *kernelT = 0.0;
        }

     } // contribution check
    
     return kernel;
}


__global__ void source_kernel(int Npart, int it, float *p0_d, float *p1_d, float *p2_d, float *p3_d, float *r0_d, float *r1_d, float *r2_d, float *r3_d, float *mi_d, float *gi_d, float *bi_d, float *Sb_d, float *St_d, float *Sx_d, float *Sy_d, float *Sn_d, float *Ttt_d, float *Ttx_d, float *Tty_d, float *Ttn_d, float *Txx_d, float *Txy_d, float *Txn_d, float *Tyy_d, float *Tyn_d, float *Tnn_d, parameters params)
{

  long int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
  long int cid = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;

  // parameters
  float sigma = params.SIGMA;
  float delta_tau = params.DELTA_TAU;
  float gmax = params.GAMMA_MAX;
  float t0 = params.T0;
  int nev = params.NEV;
  int Ntot = params.NTOT;
  int Nx = params.NX;
  int Ny = params.NY;
  int Nn = params.NN;
  float dt = params.DT;
  float dx = params.DX;
  float dy = params.DY;
  float dn = params.DN;
  
  // common factors
  float sigma2 = sigma*sigma;
  float SigInv = 1.0/(2.0*sigma2);
  float prefac = 1.0/pow(2.0*PI*sigma2,1.5); //[1/fm^3]
    
  float d_tauInv = 1.0/delta_tau;
  float prefactor = 0.5 * d_tauInv * prefac;
    
  float nevInv = 1.0/(float)nev;

  float facB = prefactor * nevInv; //[1/fm^4]
  float facT = prefactor * nevInv / hbarc; //[1/(GeV*fm^5)], momentum from UrQMD is in [GeV], but in Hydro code we need sources in [fm]
  float facTensor = prefac * nevInv / hbarc; //[1/(GeV*fm^4)]
    
  // gamma regulation
  float gmax2 = gmax * gmax;
  float vmax = sqrt(1-1/gmax2);
    
  // proper time
  float tau = t0 + ((float)it) * dt;
  float tauInv = 1/tau;

  
  //==========================================================================
  // loop over all cells (x, y, eta)
    
  if (cid < Ntot)
  {
        // reconstruct indices manually using
        int k = cid / (Nx * Ny);
        int j = ( cid - (k * Nx * Ny) ) / Nx;
        int i = cid - (k * Nx * Ny) - (j * Nx);

        // space-time where to calculate sources
        float r0 = tau;
        float r1 = ((float)i - ((float)Nx - 1.0)/2.0) * dx;
        float r2 = ((float)j - ((float)Ny - 1.0)/2.0) * dy;
        float r3 = ((float)k - ((float)Nn - 1.0)/2.0) * dn;
      
        // coordinate transformation matrix element
        float cheta = cosh(r3); // cosh(eta_s)
        float sheta = sinh(r3); // sinh(eta_s)
      
        // calculate the source terms for rid ijk and write them in the output file
        float Sb = 0.0;
        float St = 0.0;
        float Sx = 0.0;
        float Sy = 0.0;
        float Sn = 0.0;
      
#ifdef INITIAL_TENSOR
        float Ttt = 0.0;
        float Ttx = 0.0;
        float Tty = 0.0;
        float Ttn = 0.0;
        float Txx = 0.0;
        float Txy = 0.0;
        float Txn = 0.0;
        float Tyy = 0.0;
        float Tyn = 0.0;
        float Tnn = 0.0;
#endif

        //==========================================================================
        // loop over all particles

        for (int m = 0; m < Npart; ++m)
        {

            // ********************************************************
            // calculate the kernel
            // ********************************************************
            
            float kernel = 0.0;
            float kernelT = 0.0;
            
            kernel = KernelCartesian(r0, r1, r2, r3, r0_d[m], r1_d[m], r2_d[m], r3_d[m], p0_d[m], p1_d[m], p2_d[m], p3_d[m],
                                     mi_d[m], gi_d[m], gmax, vmax, sigma2, delta_tau, d_tauInv, SigInv, &kernelT);
            
            // ********************************************************
            // calculate the source
            // ********************************************************
            
            float pm0 = cheta * p0_d[m] - sheta * p3_d[m];
            float pm3 = (-sheta * p0_d[m] + cheta * p3_d[m]) * tauInv;

            Sb = Sb + kernel * bi_d[m]; // [1]
            St = St + kernel * pm0;     // [GeV]
            Sx = Sx + kernel * p1_d[m]; // [GeV]
            Sy = Sy + kernel * p2_d[m]; // [GeV]
            Sn = Sn + kernel * pm3;     // [GeV/fm]
            
#ifdef INITIAL_TENSOR
            float ptauInv = 1 / p0_d[m]; // p0_d[m] is pt, not p^tau
            Ttt = Ttt + kernelT * ptauInv * pm0 * pm0; // kernelT is unitless
            Ttx = Ttx + kernelT * ptauInv * pm0 * p1_d[m]; // [GeV]
            Tty = Tty + kernelT * ptauInv * pm0 * p2_d[m];
            Ttn = Ttn + kernelT * ptauInv * pm0 * pm3;
            Txx = Txx + kernelT * ptauInv * p1_d[m] * p1_d[m];
            Txy = Txy + kernelT * ptauInv * p1_d[m] * p2_d[m];
            Txn = Txn + kernelT * ptauInv * p1_d[m] * pm3;
            Tyy = Tyy + kernelT * ptauInv * p2_d[m] * p2_d[m];
            Tyn = Tyn + kernelT * ptauInv * p2_d[m] * pm3;
            Tnn = Tnn + kernelT * ptauInv * pm3 * pm3; // [GeV/fm^2] = [1/GeV] * [GeV/fm] * [GeV/fm]
#endif
        } //for (int m = 0; m < N; ++m)

        //==========================================================================
        // Write the source terms to arrays
      
        Sb_d[cid] = facB * Sb; // [1/fm^4] = [1/fm^4] * [1]
        St_d[cid] = facT * St; // [1/fm^5] = [1/(fm^5*GeV)] * [GeV]
        Sx_d[cid] = facT * Sx; // [1/fm^5] = [1/(fm^5*GeV)] * [GeV]
        Sy_d[cid] = facT * Sy; // [1/fm^5] = [1/(fm^5*GeV)] * [GeV]
        Sn_d[cid] = facT * Sn; // [1/fm^6] = [1/(fm^5*GeV)] * [GeV/m]
#ifdef INITIAL_TENSOR
        Ttt_d[cid] = facTensor * Ttt; // facTensor = [1/(GeV*fm^4)]
        Ttx_d[cid] = facTensor * Ttx; // [1/fm^4] = [1/(GeV*fm^4)] * [GeV]
        Tty_d[cid] = facTensor * Tty;
        Ttn_d[cid] = facTensor * Ttn;
        Txx_d[cid] = facTensor * Txx;
        Txy_d[cid] = facTensor * Txy;
        Txn_d[cid] = facTensor * Txn;
        Tyy_d[cid] = facTensor * Tyy;
        Tyn_d[cid] = facTensor * Tyn;
        Tnn_d[cid] = facTensor * Tnn; // [1/fm^6] = [1/(GeV*fm^4)] * [GeV/fm^2]
#endif
   } //if (cid < Ntot)
}
