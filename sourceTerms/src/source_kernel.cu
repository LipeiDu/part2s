#include "hip/hip_runtime.h"

#include "parameters.h"

__global__ void source_kernel(int Npart, int it, float *p0_d, float *p1_d, float *p2_d, float *p3_d, float *r0_d, float *r1_d, float *r2_d, float *r3_d,
                              float *mi_d, float *gi_d, float *bi_d, float *Sb_d, float *St_d, float *Sx_d, float *Sy_d, float *Sn_d, parameters params)
{

  long int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
  long int tid = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;

  // parameters
  int Ntot = params.NTOT;
  int nev = params.NEV;
  float sigma = params.SIGMA;
  float delta_tau = params.DELTA_TAU;
  float gmax = params.GAMMA_MAX;
  float t0 = params.T0;
  int Nx = params.NX;
  int Ny = params.NY;
  int Nn = params.NN;
  float dt = params.DT;
  float dx = params.DX;
  float dy = params.DY;
  float dn = params.DN;
  
  // common factors
  float sigma2 = sigma*sigma;
  float SigInv = 1.0/(2.0*sigma2);
  float prefac = 1.0/pow(2.0*PI*sigma2,1.5);
    
  float d_tauInv = 1.0/delta_tau;
  float prefactor = 0.5 * d_tauInv * prefac;
    
  float nevInv = 1.0/(float)nev;

  float facB = prefactor * nevInv; //[1/fm^4]
  float facT = prefactor * nevInv / hbarc; //[1/(GeV*fm^5)]
    
  float tau = t0 + ((float)it) * dt;
  float tau2 = tau * tau;
  float tauInv = 1/tau;
    
  float gmax2 = gmax * gmax;
  float vmax = sqrt(1-1/gmax2);
  
  //==========================================================================
  // loop over all cells
    
  if (tid < Ntot)
  {
    // reconstruct indices manually using
    int k = tid / (Nx * Ny);
    int j = ( tid - (k * Nx * Ny) ) / Nx;
    int i = tid - (k * Nx * Ny) - (j * Nx);

    // space-time where to calculate sources
    float r0 = tau;
    float r1 = ((float)i - ((float)Nx - 1.0)/2.0) * dx;
    float r2 = ((float)j - ((float)Ny - 1.0)/2.0) * dy;
    float r3 = ((float)k - ((float)Nn - 1.0)/2.0) * dn;
      
    // coordinate transformation matrix element
    float cheta = cosh(r3); // cosh(eta_s)
    float sheta = sinh(r3); // sinh(eta_s)
      
    // calculate the source terms for rid ijk and write them in the output file
    float Sb = 0.0;
    float St = 0.0;
    float Sx = 0.0;
    float Sy = 0.0;
    float Sn = 0.0;

    //==========================================================================
    // loop over all particles

    for (int m = 0; m < Npart; ++m)
    {
        // I. if the contraction factor is too large, rescale it
        
        float gratio = 1.0;
        float vratio = 1.0;
        
        // now the momentum is still in Cartesian coordinates
        float mInv = 1/mi_d[m];
        float u0 = p0_d[m] * mInv;
        float u1 = p1_d[m] * mInv;
        float u2 = p2_d[m] * mInv;
        float u3 = p3_d[m] * mInv; // u3 doesn't have 1/tau
        
        if (gi_d[m] > gmax)
        {
            gratio = gmax / gi_d[m];
            float gi2 = gi_d[m] * gi_d[m];
            float vi = sqrt(1-1/gi2);
            vratio = vmax / vi;
            //printf("gi_d[m]=%f, gratio=%f\n",gi_d[m],gratio);
            
            float ratio = gratio * vratio;

            u0 = u0 * gratio;
            u1 = u1 * ratio;
            u2 = u2 * ratio;
            u3 = u3 * ratio;
         }
        
        // II. calculate the kernel
        
        float b_i = bi_d[m];
        
        float d0 = r0 - r0_d[m];
        float d1 = r1 - r1_d[m];
        float d2 = r2 - r2_d[m];
        float d3 = r3 - r3_d[m];
        
        // conversion into Milne
        
        float um0 = cheta * u0 - sheta * u3;
        float um3 = -sheta * u0 + cheta * u3; // attention: no 1/tau here
        
        //printf("norm of u = %f\n", um0 * um0 - u1 * u1 - u2 * u2 - um3 * um3);
        //printf("p2-m2 = %f\n", p0_d[m] * p0_d[m] - p1_d[m] * p1_d[m] - p2_d[m] * p2_d[m] - p3_d[m] * p3_d[m] - mi_d[m] * mi_d[m]);

        float dxsqd = d0 * d0 - d1 * d1 - d2 * d2 - tau2 * d3 * d3;
        float udotx = um0 * d0 - u1 * d1 - u2 * d2 - um3 * d3;
        float dist4d = dxsqd - udotx * udotx;
        
        //if (dist4d <= 9 * sigma2 && udotx <= 3 * delta_tau) // if the particle's contribution is important
        //{
        // Smearing kernel
        float exponent = dist4d * SigInv;
        float exponentiation = exp(exponent);
        
        float uxnorm = udotx * d_tauInv;
        float ch = cosh(uxnorm);
        float chInv = 1/ch;
        float kernel = chInv * chInv * exponentiation; // [1], only the cosh^2 and exp product
        
        // III. calculate the source
        if (isnan(kernel)) printf("Kernel is nan for particle %d\n",m);
            
        if (!isnan(kernel))// if kernel is nan for some reasons, skip this particle
        {
            // momentum is not rescale to conserve energy-momentum, into Milne
            float pm0 = cheta * p0_d[m] - sheta * p3_d[m];
            float pm3 = -sheta * p0_d[m] + cheta * p3_d[m];
            
            //printf("p2-m2 = %f\n", pm0 * pm0 - p1_d[m] * p1_d[m] - p2_d[m] * p2_d[m] - pm3 * pm3 - mi_d[m] * mi_d[m]);
            
            Sb = Sb + b_i * prefac * exponentiation * gi_d[m];
            //Sb = Sb + kernel * b_i * gi_d[m] * gratio;     // [1]
            St = St + kernel * pm0;     // [GeV]
            Sx = Sx + kernel * p1_d[m]; // [GeV]
            Sy = Sy + kernel * p2_d[m]; // [GeV]
            Sn = Sn + kernel * pm3;     // [GeV], pm3 is [GeV] by defination above
         }
        
        //} // contribution check
        
     } //for (int m = 0; m < N; ++m)

    //==========================================================================
    // Write the source terms to arrays
      Sb_d[tid] = Sb;
    //Sb_d[tid] = facB * Sb; // [1/fm^4] = [1/fm^4] * [1]
    St_d[tid] = facT * St; // [1/fm^5] = [1/(fm^5*GeV)] * [GeV]
    Sx_d[tid] = facT * Sx; // [1/fm^5] = [1/(fm^5*GeV)] * [GeV]
    Sy_d[tid] = facT * Sy; // [1/fm^5] = [1/(fm^5*GeV)] * [GeV]
    Sn_d[tid] = facT * Sn * tauInv; // [1/fm^6] = [1/(fm^5*GeV)] * [GeV] * [1/fm]
      
   } //if (tid < Ntot)
}
