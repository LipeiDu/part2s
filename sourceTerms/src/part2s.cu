#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////
//               DynamicalSource.cpp                  //
//                                                    //
//         Created by Lipei Du on 7/23/18.            //
////////////////////////////////////////////////////////

#include <stdio.h>
#include <iostream>
#include <istream>
#include <fstream>
#include <math.h>
#include <string>
#include <iomanip>
#include "parameters.h"
#include "source_kernel.cu"
#include <H5Cpp.h>
#include <H5File.h>

using namespace std;

////////////////////////////////////////////////////////////////////////////
//                            Parameters reader                           //
////////////////////////////////////////////////////////////////////////////

void readInParameters(struct parameters &params)
{
    char dummyChar[255];
    int dummyInt;
    float dummyFloat;

    FILE *fileIn;
    char fname[255];
    sprintf(fname, "parameter.dat");
    fileIn = fopen(fname, "r");

    if (fileIn == NULL)
    {
        printf("Couldn't open parameter.dat . Using default values!\n");
    }
    else
    {
        fscanf(fileIn, "%s\t%d\n", dummyChar, &dummyInt);
        params.NEV = dummyInt;
        fscanf(fileIn, "%s\t%f\n", dummyChar, &dummyFloat);
        params.SIGMA = dummyFloat;
        fscanf(fileIn, "%s\t%f\n", dummyChar, &dummyFloat);
        params.GAMMA_MAX = dummyFloat;
        fscanf(fileIn, "%s\t%f\n", dummyChar, &dummyFloat);
        params.DELTA_TAU = dummyFloat;
        fscanf(fileIn, "%s\t%f\n", dummyChar, &dummyFloat);
        params.TAUFORM = dummyFloat;
        fscanf(fileIn, "%s\t%f\n", dummyChar, &dummyFloat);
        params.T0 = dummyFloat;
        fscanf(fileIn, "%s\t%d\n", dummyChar, &dummyInt);
        params.NX = dummyInt;
        fscanf(fileIn, "%s\t%d\n", dummyChar, &dummyInt);
        params.NY = dummyInt;
        fscanf(fileIn, "%s\t%d\n", dummyChar, &dummyInt);
        params.NN = dummyInt;
        fscanf(fileIn, "%s\t%d\n", dummyChar, &dummyInt);
        params.NT = dummyInt;
        fscanf(fileIn, "%s\t%f\n", dummyChar, &dummyFloat);
        params.DT = dummyFloat;
        fscanf(fileIn, "%s\t%f\n", dummyChar, &dummyFloat);
        params.DX = dummyFloat;
        fscanf(fileIn, "%s\t%f\n", dummyChar, &dummyFloat);
        params.DY = dummyFloat;
        fscanf(fileIn, "%s\t%f\n", dummyChar, &dummyFloat);
        params.DN = dummyFloat;
    }

    fclose(fileIn);
}


int main()
{
  printf("\n\n\n");
  printf("++++++++++++++Program started+++++++++++++++\n");

  ////////////////////////////////////////////////////////////////////////////
  //                            Initialize parameters                       //
  ////////////////////////////////////////////////////////////////////////////


  // declare parameters struct
  struct parameters params;

  // default values
  params.NEV = 1;
  params.SIGMA = 1.0;
  params.GAMMA_MAX = 2.0;
  params.DELTA_TAU = 0.5;
  params.TAUFORM = 0.2;
  params.T0 = 0.5;
  params.NX = 131;
  params.NY = 131;
  params.NN = 61;
  params.NT = 80;
  params.DT = 0.05;
  params.DX = 0.15;
  params.DY = 0.15;
  params.DN = 0.15;

  // read in chosen parameters from parameters.dat if such a file exists
  readInParameters(params);

  params.NTOT = (params.NX * params.NY * params.NN);

  // pass values
  float tauform = params.TAUFORM;
  int Nx = params.NX;
  int Ny = params.NY;
  int Nn = params.NN;
  int Nt = params.NT;
  int Ntot = params.NTOT;
  float t0 = params.T0;
  float dt = params.DT;
  float dx = params.DX;
  float dy = params.DY;
  float dn = params.DN;

  ////////////////////////////////////////////////////////////////////////////
  //                            Process all event files                     //
  ////////////////////////////////////////////////////////////////////////////
    
  // In this part, loop over all events; for each event, calculate the coordinates after formation time,
  // calculate the center and angle, recenter and rotate; write the coordinates in Milen in AllSets.dat;
    
  int Npart = 0; // total number of particles in all event files to be averaged over
  int Nbtot = 0;
    
  FILE *allsetfile;
  char allsetname[255];
  sprintf(allsetname, "%s.dat", "output/AllSets"); // All set.dat will be written in this file after recentering and rotation
  allsetfile = fopen(allsetname, "w");

  // loop over all events
  for (int iev=1; iev<params.NEV+1; ++iev)
  {
      
      printf("**********Processing Set%d.dat**********\n", iev);

      FILE *eventfile;
      char eventname[255];
      sprintf(eventname, "%s%d.dat", "Set",iev);
      eventfile = fopen(eventname, "r");

      ////////////////////////////////////////////////////////////////////////////
      //                             Converting coordinates                     //
      ////////////////////////////////////////////////////////////////////////////
      
      // read in the particle list from UrQMD; get the info. in Milne.
      
      printf("Converting into Milne...\n");
      
      FILE *milnefile;
      char milnename[255];
      sprintf(milnename, "%s%d.dat", "output/Milne", iev);
      milnefile = fopen(milnename, "w");
      
      float r_i[4], p_i[4];
      float rm_i[4];
      
      for (int j=0; j<4; ++j)
      {
          r_i[j] = 0;
          p_i[j] = 0;
          rm_i[j] = 0;
      }
      
      float m_i = 0;
      float tform_i = 0;
      float b_i = 0;
      float g_i = 0;// gamma factor
      
      int NpartEv = 0; // total number of particles in this event
      
      // to get the center of the energy distribution
      float Etotx = 0.0;
      float Etoty = 0.0;
      float Etot = 0.0;

      if(eventfile==NULL){
        printf("Set%d.dat could not be opened...\n", iev);
      }
      else
      {
        fseek(eventfile,0L,SEEK_SET);
        fscanf(eventfile,"%*[^\n]%*c");//Skip the header line

        while(!feof(eventfile))
        {

            // read in particle list from UrQMD
            fscanf(eventfile, "%f %f %f %f %f %f %f %f %f %f %f\n", &r_i[0], &r_i[1], &r_i[2], &r_i[3], &p_i[0], &p_i[1], &p_i[2], &p_i[3], &m_i, &tform_i, &b_i);

            // gamma factor of particle i
            g_i = p_i[0]/m_i;

            // calculate the final postion of each particle after the formation time

            float tform = tauform * g_i;
            float tformE = tform/p_i[0];

            r_i[0] = r_i[0] + tform;
            r_i[1] = r_i[1] + p_i[1] * tformE;
            r_i[2] = r_i[2] + p_i[2] * tformE;
            r_i[3] = r_i[3] + p_i[3] * tformE;

            // transfer into Milne coordinates

            rm_i[0] = sqrt(r_i[0]*r_i[0]-r_i[3]*r_i[3]); // tau
            rm_i[1] = r_i[1]; // x
            rm_i[2] = r_i[2]; // y
            rm_i[3] = 0.5 * log((r_i[0]+r_i[3])/(r_i[0]-r_i[3]+1.e-30)); // eta_s


            // write Milne in output file (WARNING: nothing is in Milne in this file, 04/05/2019)
            // skip particles outside light cone
            if ( isnan(rm_i[0]) || isnan(rm_i[3]) )
            {
              printf("*Warning* : found particle outside light cone (excluding it...)\n");
            }
            else
            {
              fprintf(milnefile,"%.8f\t%.8f\t%.8f\t%.8f\t%.8f\t%.8f\t%.8f\t%.8f\t%.8f\t%.8f\t%.8f\n",
                      r_i[0],r_i[1],r_i[2],r_i[3],p_i[0],p_i[1],p_i[2],p_i[3],m_i,g_i,b_i);

              NpartEv++;
              Nbtot = Nbtot + b_i;

              // center of the energy distribution
              Etot = Etot + p_i[0];
              Etotx =  Etotx + p_i[0] * r_i[1];
              Etoty =  Etoty + p_i[0] * r_i[2];
            }
        }

        fclose(eventfile);
        fclose(milnefile);

        printf("Total number of particles in Set%d.dat is %d.\n", iev, NpartEv);
      }
      
      Npart = Npart + NpartEv;

      
      ////////////////////////////////////////////////////////////////////////////
      //                             Find CM and plane angle                    //
      ////////////////////////////////////////////////////////////////////////////
      
      // The center has almost been found at this point, but the angle needs to be calculated with respect to the center.
      // So the particle list needs to be looped over again

      // Section a: center
      
      float CMx = 0.0;
      float CMy = 0.0;

      CMx = Etotx/Etot;
      CMy = Etoty/Etot;

      printf("Center of Mass is: CMx= %.3f, CMy= %.3f.\n", CMx, CMy);

      for (int j=0; j<4; ++j)
      {
        rm_i[j] = 0;
        p_i[j] = 0;
      }
      
      m_i = 0;
      g_i = 0;
      b_i = 0;

      // to get the center of the energy distribution
      float psi = 0.0;
      float avgxy = 0.0;
      float avgy2x2 = 0.0;
      
      // Section b: participant plane angle
      
      milnefile = fopen(milnename, "r");
      
      if(milnefile==NULL){
          printf("Milne%d.dat could not be opened...\n", iev);
      }
      else{
          fseek(milnefile,0L,SEEK_SET);
          
          while(!feof(milnefile))
          {
              // write particle info, x in Milne, p in Cartesian
              fscanf(milnefile, "%f %f %f %f %f %f %f %f %f %f %f\n", &rm_i[0], &rm_i[1], &rm_i[2], &rm_i[3], &p_i[0], &p_i[1], &p_i[2], &p_i[3], &m_i, &g_i, &b_i);
              
              avgxy =  avgxy + p_i[0] * (rm_i[1]-CMx) * (rm_i[2]-CMy);
              avgy2x2 =  avgy2x2 + p_i[0] * ((rm_i[1]-CMx)*(rm_i[1]-CMx) - (rm_i[2]-CMy)*(rm_i[2]-CMy));
          }
      }
      
      // participant plane angle
      psi = 0.5 * atan (2 * avgxy/(avgy2x2+1.e-30));
      printf("Participant plane angle is: %.3f rad or %.3f degree.\n", psi, psi*180/3.1415);
      
      fclose(milnefile);


      ////////////////////////////////////////////////////////////////////////////
      //                             Recenter and rotate                        //
      ////////////////////////////////////////////////////////////////////////////

      float r0 = 0.0;
      float r1 = 0.0;
      float r2 = 0.0;
      float r3 = 0.0;
      float p0 = 0.0;
      float p1 = 0.0;
      float p2 = 0.0;
      float p3 = 0.0;
      float mi = 0.0;
      float gi = 0.0;
      float bi = 0.0;
    
      milnefile = fopen(milnename, "r");
      
      if(milnefile==NULL){
          printf("Milne%d.dat could not be opened...\n", iev);
      }
      else{
          fseek(milnefile,0L,SEEK_SET);
          
          while(1)
          {
              fscanf(milnefile,"%e %e %e %e %e %e %e %e %e %e %e", &r0, &r1, &r2, &r3, &p0, &p1, &p2, &p3, &mi, &gi, &bi);
              if(feof(milnefile)) break;
              
              // recenter and rotate
              
              r1 = r1 - CMx;
              r2 = r2 - CMy;
              float xp =  r1 * cos(psi) + r2 * sin(psi);
              float yp = -r1 * sin(psi) + r2 * cos(psi);
              
              float p1p =  p1 * cos(psi) + p2 * sin(psi);
              float p2p = -p1 * sin(psi) + p2 * cos(psi);
              
              fprintf(allsetfile,"%.8f\t%.8f\t%.8f\t%.8f\t%.8f\t%.8f\t%.8f\t%.8f\t%.8f\t%.8f\t%.8f\n",r0,xp,yp,r3,p0,p1p,p2p,p3,mi,gi,bi);
          }
      }
      
      fclose(milnefile);
  }
    
  fclose(allsetfile);
    
  printf("***************************************\n");
  printf("Total number of particles in All Sets is %d, net baryon number is %d.\n", Npart, Nbtot);

  ////////////////////////////////////////////////////////////////////////////
  //                             All particles Milne                        //
  ////////////////////////////////////////////////////////////////////////////

  // read in all the particles in all events in Cartesian

  float r0[Npart], r1[Npart], r2[Npart], r3[Npart];
  float p0[Npart], p1[Npart], p2[Npart], p3[Npart];
  float mi[Npart], gi[Npart], bi[Npart];

  for (int i = 0; i < Npart; ++i)
  {
    r0[i] = 0.0;
    r1[i] = 0.0;
    r2[i] = 0.0;
    r3[i] = 0.0;
    p0[i] = 0.0;
    p1[i] = 0.0;
    p2[i] = 0.0;
    p3[i] = 0.0;
    mi[i] = 0.0;
    gi[i] = 0.0;
    bi[i] = 0.0;
  }

  allsetfile = fopen(allsetname, "r");

  if (allsetfile == NULL)
    printf("The particle list in AllSets.dat couldn't be opened...\n");
  else
  {
    fseek(allsetfile,0L,SEEK_SET);
    for (int i = 0; i < Npart; ++i)
    {
        fscanf(allsetfile,"%e %e %e %e %e %e %e %e %e %e %e", &r0[i], &r1[i], &r2[i], &r3[i], &p0[i], &p1[i], &p2[i], &p3[i], &mi[i], &gi[i], &bi[i]);
    }
  }
  fclose(allsetfile);

  ////////////////////////////////////////////////////////////////////////////
  //                             Source terms                               //
  ////////////////////////////////////////////////////////////////////////////

  printf("Calculating source terms...\n");
  hipDeviceSynchronize();
  hipError_t err;

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("Error at very beginning: %s\n", hipGetErrorString(err));
    err = hipSuccess;
  }

  //declare and allocate device arrays to hold particle info from UrQMD
  float *p0_d, *p1_d, *p2_d, *p3_d;
  float *r0_d, *r1_d, *r2_d, *r3_d;
  float *mi_d, *gi_d, *bi_d;
    
  hipMalloc( (void**) &p0_d, Npart * sizeof(float) );
  hipMalloc( (void**) &p1_d, Npart * sizeof(float) );
  hipMalloc( (void**) &p2_d, Npart * sizeof(float) );
  hipMalloc( (void**) &p3_d, Npart * sizeof(float) );
  hipMalloc( (void**) &r0_d, Npart * sizeof(float) );
  hipMalloc( (void**) &r1_d, Npart * sizeof(float) );
  hipMalloc( (void**) &r2_d, Npart * sizeof(float) );
  hipMalloc( (void**) &r3_d, Npart * sizeof(float) );
  hipMalloc( (void**) &mi_d, Npart * sizeof(float) );
  hipMalloc( (void**) &gi_d, Npart * sizeof(float) );
  hipMalloc( (void**) &bi_d, Npart * sizeof(float) );

  //declare and allocate device source term arrays
  float *Sb_d, *St_d, *Sx_d, *Sy_d, *Sn_d;
  hipMalloc( (void**) &Sb_d, Ntot * sizeof(float) );
  hipMalloc( (void**) &St_d, Ntot * sizeof(float) );
  hipMalloc( (void**) &Sx_d, Ntot * sizeof(float) );
  hipMalloc( (void**) &Sy_d, Ntot * sizeof(float) );
  hipMalloc( (void**) &Sn_d, Ntot * sizeof(float) );


  err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("Error in device memory allocation: %s\n", hipGetErrorString(err));
    err = hipSuccess;
  }

  // copy input arrays from host to device
  hipMemcpy( p0_d, p0, Npart * sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( p1_d, p1, Npart * sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( p2_d, p2, Npart * sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( p3_d, p3, Npart * sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( r0_d, r0, Npart * sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( r1_d, r1, Npart * sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( r2_d, r2, Npart * sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( r3_d, r3, Npart * sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( mi_d, mi, Npart * sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( gi_d, gi, Npart * sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( bi_d, bi, Npart * sizeof(float), hipMemcpyHostToDevice );

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("Error in a hipMemcpy: %s\n", hipGetErrorString(err));
    err = hipSuccess;
  }

  //zero the device source arrays first
  hipMemset( Sb_d, 0.0, Ntot * sizeof(float));
  hipMemset( St_d, 0.0, Ntot * sizeof(float));
  hipMemset( Sx_d, 0.0, Ntot * sizeof(float));
  hipMemset( Sy_d, 0.0, Ntot * sizeof(float));
  hipMemset( Sn_d, 0.0, Ntot * sizeof(float));

  //kernel launch parameters
  int threadsX = 16;
  int threadsY = 16;
  int threadsZ = 1;
  int blocksX = (Nx+threadsX-1)/threadsX;
  int blocksY = (Ny+threadsY-1)/threadsY;
  int blocksZ = (Nn+threadsZ-1)/threadsZ;
    
  printf("CUDA kernel parameters:\n");
  printf("dim3 grids = ( %d, %d, %d )\n", blocksX, blocksY, blocksZ);
  printf("dim3 threads = ( %d, %d, %d )\n", threadsX, threadsY, threadsZ);
    
  dim3 grids( blocksX, blocksY, blocksZ );
  dim3 threads( threadsX, threadsY, threadsZ);

  //host arrays for source terms
  float *Sb, *St, *Sx, *Sy, *Sn;
  Sb = (float *)calloc( Ntot, sizeof(float) );
  St = (float *)calloc( Ntot, sizeof(float) );
  Sx = (float *)calloc( Ntot, sizeof(float) );
  Sy = (float *)calloc( Ntot, sizeof(float) );
  Sn = (float *)calloc( Ntot, sizeof(float) );

  //an array to hold all info for all the source terms compressed to 1d for hdf5 writer
  float *Sall;
  Sall = (float *)calloc( 5*Ntot, sizeof(float) );
    
  ////////////////////////////////////////////////////////////////////////////
  //                             Calculation all time steps                 //
  ////////////////////////////////////////////////////////////////////////////

  FILE *sourcefile;
  char source_fname[255];
  char finame[255];
    
  float Norm = 0.0;// test the normalization of the kernel, add up all time steps

  //loop over time steps, calling kernel for each and writing to file
  for (int n = 1; n < Nt+1; ++n)
  {
    if ( (n % 10) == 1 ) printf("Calculating source term for n = %d of %d\n", n, Nt);
      
    int it = n-1;
    float tau = t0 + ((float)it) * dt;
    
    printf("tau = %f\n", tau);
      
    sprintf(source_fname, "%s%d.h5", "output/Sources", n);

    source_kernel<<< grids, threads >>>(Npart, it, p0_d, p1_d, p2_d, p3_d, r0_d, r1_d, r2_d, r3_d, mi_d, gi_d, bi_d, Sb_d, St_d, Sx_d, Sy_d, Sn_d, params);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
      printf("Error in source kernel: %s\n", hipGetErrorString(err));
      err = hipSuccess;
    }
    
    //now copy results from device to host
    hipMemcpy( Sb, Sb_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( St, St_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( Sx, Sx_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( Sy, Sy_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( Sn, Sn_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );

    //compress all data into the 1d array to pass to hdf5 writer
    for (int is = 0; is < Ntot; is++)
    {
      Sall[is] = Sb[is];
      Sall[Ntot + is] = St[is];
      Sall[2 * Ntot + is] = Sx[is];
      Sall[3 * Ntot + is] = Sy[is];
      Sall[4 * Ntot + is] = Sn[is];
    }

    //printf("Writing source terms to file...\n\n");
    H5::H5File file(source_fname, H5F_ACC_TRUNC);
      
    // dataset dimensions
    hsize_t dimsf[4];
    dimsf[0] = Nx;
    dimsf[1] = Ny;
    dimsf[2] = Nn;
    dimsf[3] = 5;

    H5::DataSpace dataspace(4, dimsf);
    H5::DataType datatype(H5::PredType::NATIVE_FLOAT);
    H5::DataSet dataset = file.createDataSet("data", datatype, dataspace);
    dataset.write(Sall, H5::PredType::NATIVE_FLOAT);

    //FOR TESTING write ascii files
    
    sprintf(finame, "%s%d.dat", "output/Sources", n);
    sourcefile = fopen(finame, "w");
      
    for (int i = 0; i < Nx; ++i)
    {
      for (int j = 0; j < Ny; ++j)
      {
        for (int k = 0; k < Nn; ++k)
        {
          
          float x   = ((float)i - ((float)Nx - 1.0)/2.0) * dx;
          float y   = ((float)j - ((float)Ny - 1.0)/2.0) * dy;
          float eta = ((float)k - ((float)Nn - 1.0)/2.0) * dn;

          int s = i + j * (Nx) + k * (Nx * Ny);

          fprintf(sourcefile, "%.8f\t%.8f\t%.8f\t%.8f\t%.8f\t%.8f\t%.8f\t%.8f\n", x, y, eta, St[s], Sx[s], Sy[s], Sn[s], Sb[s]);
            
            Norm = Norm + Sb[s] * tau;
        } // for (int k )
      } //for (int j)
    } //for (int i )
    fclose(sourcefile);
      
    printf("Numerical Nb = %lf\n",Norm * dt * dx * dy * dn);
    //Norm = 0.0;
    //FOR TESTING write ascii files
  } // for (int n )
    
  Norm = Norm * dt * dx * dy * dn;
  printf("Numerical Nb = %lf\n",Norm);
    
  ////////////////////////////////////////////////////////////////////////////
  //                             Clean up                                   //
  ////////////////////////////////////////////////////////////////////////////
    
  printf("Freeing memory\n");
  
  free(Sb);
  free(St);
  free(Sx);
  free(Sy);
  free(Sn);

  hipFree(p0_d);
  hipFree(p1_d);
  hipFree(p2_d);
  hipFree(p3_d);
  hipFree(r0_d);
  hipFree(r1_d);
  hipFree(r2_d);
  hipFree(r3_d);
  hipFree(mi_d);
  hipFree(gi_d);
  hipFree(bi_d);
  hipFree(Sb_d);
  hipFree(St_d);
  hipFree(Sx_d);
  hipFree(Sy_d);
  hipFree(Sn_d);

  printf("Done. Goodbye! \n");
}
