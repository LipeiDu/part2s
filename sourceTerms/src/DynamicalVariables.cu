#include "ParameterReader.cpp"
#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////
// device arrays
////////////////////////////////////////////////////////////////////////////

//declare and allocate device arrays to hold particle info from UrQMD
extern float *p0_d, *p1_d, *p2_d, *p3_d;
extern float *r0_d, *r1_d, *r2_d, *r3_d;
extern float *mi_d, *gi_d, *bi_d;

//declare and allocate device source term arrays and tensors
extern float *Sb_d, *St_d, *Sx_d, *Sy_d, *Sn_d;
extern float *Ttt_d, *Ttx_d, *Tty_d, *Ttn_d, *Txx_d, *Txy_d, *Txn_d, *Tyy_d, *Tyn_d, *Tnn_d;
extern float *Nt_d, *Nx_d, *Ny_d, *Nn_d;

////////////////////////////////////////////////////////////////////////////
// host arrays
////////////////////////////////////////////////////////////////////////////

//host arrays for source terms
extern float *Sb, *St, *Sx, *Sy, *Sn;

//an array to hold all info for all the source terms compressed to 1d for hdf5 writer
extern float *Sall;

//host arrays for tensor
extern float **stressTensor, **shearTensor, **flowVelocity;
extern float *energyDensity, *pressure, *temperature, *bulkPressure;
extern float **baryonCurrent, **baryonDiffusion, *baryonDensity;

//arrays for hdf5 writer
extern float *stressAll;
extern float *shearAll;
extern float *primaryAll;
extern float *baryonAll;

////////////////////////////////////////////////////////////////////////////
// allocation
////////////////////////////////////////////////////////////////////////////

float *p0_d, *p1_d, *p2_d, *p3_d;
float *r0_d, *r1_d, *r2_d, *r3_d;
float *mi_d, *gi_d, *bi_d;

float *Sb_d, *St_d, *Sx_d, *Sy_d, *Sn_d;
float *Ttt_d, *Ttx_d, *Tty_d, *Ttn_d, *Txx_d, *Txy_d, *Txn_d, *Tyy_d, *Tyn_d, *Tnn_d;
float *Nt_d, *Nx_d, *Ny_d, *Nn_d;

float *Sb, *St, *Sx, *Sy, *Sn;
float *Sall;

float **stressTensor, **shearTensor, **flowVelocity;
float *energyDensity, *pressure, *temperature, *bulkPressure;
float **baryonCurrent, **baryonDiffusion, *baryonDensity;

float *stressAll;
float *shearAll;
float *primaryAll;
float *baryonAll;

////////////////////////////////////////////////////////////////////////////

void allocateDeviceMemory(int Npart, int Ntot, hipError_t err){
    
    hipMalloc( (void**) &p0_d, Npart * sizeof(float) );
    hipMalloc( (void**) &p1_d, Npart * sizeof(float) );
    hipMalloc( (void**) &p2_d, Npart * sizeof(float) );
    hipMalloc( (void**) &p3_d, Npart * sizeof(float) );
    hipMalloc( (void**) &r0_d, Npart * sizeof(float) );
    hipMalloc( (void**) &r1_d, Npart * sizeof(float) );
    hipMalloc( (void**) &r2_d, Npart * sizeof(float) );
    hipMalloc( (void**) &r3_d, Npart * sizeof(float) );
    hipMalloc( (void**) &mi_d, Npart * sizeof(float) );
    hipMalloc( (void**) &gi_d, Npart * sizeof(float) );
    hipMalloc( (void**) &bi_d, Npart * sizeof(float) );
    
    hipMalloc( (void**) &Sb_d, Ntot * sizeof(float) );
    hipMalloc( (void**) &St_d, Ntot * sizeof(float) );
    hipMalloc( (void**) &Sx_d, Ntot * sizeof(float) );
    hipMalloc( (void**) &Sy_d, Ntot * sizeof(float) );
    hipMalloc( (void**) &Sn_d, Ntot * sizeof(float) );
    
    //zero the device source arrays first
    hipMemset( Sb_d, 0.0, Ntot * sizeof(float));
    hipMemset( St_d, 0.0, Ntot * sizeof(float));
    hipMemset( Sx_d, 0.0, Ntot * sizeof(float));
    hipMemset( Sy_d, 0.0, Ntot * sizeof(float));
    hipMemset( Sn_d, 0.0, Ntot * sizeof(float));
    
#ifdef INITIAL_TENSOR
    //declare and allocate device tensor arrays
    hipMalloc( (void**) &Ttt_d, Ntot * sizeof(float) );
    hipMalloc( (void**) &Ttx_d, Ntot * sizeof(float) );
    hipMalloc( (void**) &Tty_d, Ntot * sizeof(float) );
    hipMalloc( (void**) &Ttn_d, Ntot * sizeof(float) );
    hipMalloc( (void**) &Txx_d, Ntot * sizeof(float) );
    hipMalloc( (void**) &Txy_d, Ntot * sizeof(float) );
    hipMalloc( (void**) &Txn_d, Ntot * sizeof(float) );
    hipMalloc( (void**) &Tyy_d, Ntot * sizeof(float) );
    hipMalloc( (void**) &Tyn_d, Ntot * sizeof(float) );
    hipMalloc( (void**) &Tnn_d, Ntot * sizeof(float) );
    
    hipMalloc( (void**) &Nt_d, Ntot * sizeof(float) );
    hipMalloc( (void**) &Nx_d, Ntot * sizeof(float) );
    hipMalloc( (void**) &Ny_d, Ntot * sizeof(float) );
    hipMalloc( (void**) &Nn_d, Ntot * sizeof(float) );
    
    //zero the device tensor arrays first
    hipMemset( Ttt_d, 0.0, Ntot * sizeof(float));
    hipMemset( Ttx_d, 0.0, Ntot * sizeof(float));
    hipMemset( Tty_d, 0.0, Ntot * sizeof(float));
    hipMemset( Ttn_d, 0.0, Ntot * sizeof(float));
    hipMemset( Txx_d, 0.0, Ntot * sizeof(float));
    hipMemset( Txy_d, 0.0, Ntot * sizeof(float));
    hipMemset( Txn_d, 0.0, Ntot * sizeof(float));
    hipMemset( Tyy_d, 0.0, Ntot * sizeof(float));
    hipMemset( Tyn_d, 0.0, Ntot * sizeof(float));
    hipMemset( Tnn_d, 0.0, Ntot * sizeof(float));
    
    hipMemset( Nt_d, 0.0, Ntot * sizeof(float));
    hipMemset( Nx_d, 0.0, Ntot * sizeof(float));
    hipMemset( Ny_d, 0.0, Ntot * sizeof(float));
    hipMemset( Nn_d, 0.0, Ntot * sizeof(float));
#endif
    
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Error in device memory allocation: %s\n", hipGetErrorString(err));
        err = hipSuccess;
    }
}

void allocateHostMemory(int Ntot){
    
    //host arrays for source terms
    Sb = (float *)calloc( Ntot, sizeof(float) );
    St = (float *)calloc( Ntot, sizeof(float) );
    Sx = (float *)calloc( Ntot, sizeof(float) );
    Sy = (float *)calloc( Ntot, sizeof(float) );
    Sn = (float *)calloc( Ntot, sizeof(float) );
    
    //an array to hold all info for all the source terms compressed to 1d for hdf5 writer
    Sall = (float *)calloc( 5*Ntot, sizeof(float) );
    
#ifdef INITIAL_TENSOR
    //host arrays for tensor
    stressTensor = (float **)calloc( 10, sizeof(float*));
    for(int i = 0; i < 10; i++)
        stressTensor[i] = (float *)calloc( Ntot, sizeof(float) );
    
    shearTensor = (float **)calloc( 10, sizeof(float*));
    for(int i = 0; i < 10; i++)
        shearTensor[i] = (float *)calloc( Ntot, sizeof(float) );
    
    flowVelocity = (float **)calloc( 4, sizeof(float*));
    for(int i = 0; i < 4; i++)
        flowVelocity[i] = (float *)calloc( Ntot, sizeof(float) );
    
    energyDensity = (float *)calloc( Ntot, sizeof(float) );
    pressure  = (float *)calloc( Ntot, sizeof(float) );
    temperature = (float *)calloc( Ntot, sizeof(float) );
    bulkPressure = (float *)calloc( Ntot, sizeof(float) );
    
    //baryon sector
    baryonCurrent = (float **)calloc( 4, sizeof(float*));
    for(int i = 0; i < 4; i++)
        baryonCurrent[i] = (float *)calloc( Ntot, sizeof(float) );
    
    baryonDiffusion = (float **)calloc( 4, sizeof(float*));
    for(int i = 0; i < 4; i++)
        baryonDiffusion[i] = (float *)calloc( Ntot, sizeof(float) );
    
    baryonDensity = (float *)calloc( Ntot, sizeof(float) );
    
    //for hdf5
    stressAll = (float *)calloc( 10*Ntot, sizeof(float) ); //Tmunu
    shearAll = (float *)calloc( 10*Ntot, sizeof(float) ); //shear stress
    primaryAll = (float *)calloc( 8*Ntot, sizeof(float) ); //energy, pressure, temperature, bulk, flow velocity
    baryonAll = (float *)calloc( 9*Ntot, sizeof(float) ); //baryon density, net current, diffusion
#endif
}

void copyHostToDeviceMemory(int Npart, int Ntot, float *r0, float *r1, float *r2, float *r3, float *p0, float *p1, float *p2, float *p3, float *mi, float *gi, float *bi, hipError_t err){
    
    hipMemcpy( p0_d, p0, Npart * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( p1_d, p1, Npart * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( p2_d, p2, Npart * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( p3_d, p3, Npart * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( r0_d, r0, Npart * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( r1_d, r1, Npart * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( r2_d, r2, Npart * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( r3_d, r3, Npart * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( mi_d, mi, Npart * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( gi_d, gi, Npart * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( bi_d, bi, Npart * sizeof(float), hipMemcpyHostToDevice );
    
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Error in a hipMemcpy: %s\n", hipGetErrorString(err));
        err = hipSuccess;
    }
}

void copyDeviceToHostMemory(int Ntot, hipError_t err){
    
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Error in source kernel: %s\n", hipGetErrorString(err));
        err = hipSuccess;
    }
    
    //now copy results from device to host
    hipMemcpy( Sb, Sb_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( St, St_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( Sx, Sx_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( Sy, Sy_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( Sn, Sn_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    
#ifdef INITIAL_TENSOR
    hipMemcpy( stressTensor[0], Ttt_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( stressTensor[1], Ttx_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( stressTensor[2], Tty_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( stressTensor[3], Ttn_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( stressTensor[4], Txx_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( stressTensor[5], Txy_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( stressTensor[6], Txn_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( stressTensor[7], Tyy_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( stressTensor[8], Tyn_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( stressTensor[9], Tnn_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    
    hipMemcpy( baryonCurrent[0], Nt_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( baryonCurrent[1], Nx_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( baryonCurrent[2], Ny_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( baryonCurrent[3], Nn_d, Ntot * sizeof(float), hipMemcpyDeviceToHost );
#endif
    
}

void freeMemory(){
    free(Sb);
    free(St);
    free(Sx);
    free(Sy);
    free(Sn);
    free(Sall);
    
    hipFree(p0_d);
    hipFree(p1_d);
    hipFree(p2_d);
    hipFree(p3_d);
    hipFree(r0_d);
    hipFree(r1_d);
    hipFree(r2_d);
    hipFree(r3_d);
    hipFree(mi_d);
    hipFree(gi_d);
    hipFree(bi_d);
    hipFree(Sb_d);
    hipFree(St_d);
    hipFree(Sx_d);
    hipFree(Sy_d);
    hipFree(Sn_d);

#ifdef INITIAL_TENSOR
    for(int i = 0; i < 10; i++)
        free(stressTensor[i]);
    free(stressTensor);
    
    for(int i = 0; i < 10; i++)
        free(shearTensor[i]);
    free(shearTensor);
    
    for(int i = 0; i < 4; i++)
        free(flowVelocity[i]);
    free(flowVelocity);
    
    free(energyDensity);
    free(pressure);
    free(temperature);
    free(bulkPressure);
    
    hipFree(Ttt_d);
    hipFree(Ttx_d);
    hipFree(Tty_d);
    hipFree(Ttn_d);
    hipFree(Txx_d);
    hipFree(Txy_d);
    hipFree(Txn_d);
    hipFree(Tyy_d);
    hipFree(Tyn_d);
    hipFree(Tnn_d);
    
    //baryon section
    for(int i = 0; i < 4; i++)
        free(baryonCurrent[i]);
    free(baryonCurrent);
    
    for(int i = 0; i < 4; i++)
        free(baryonDiffusion[i]);
    free(baryonDiffusion);
    
    free(baryonDensity);

    hipFree(Nt_d);
    hipFree(Nx_d);
    hipFree(Ny_d);
    hipFree(Nn_d);
    
    // for hdf5
    free(stressAll);
    free(shearAll);
    free(primaryAll);
    free(baryonAll);
#endif
}
