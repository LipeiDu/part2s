#include "hip/hip_runtime.h"

#include "parameters.h"

__global__ void source_kernel(int Npart, int it,
                              float *p0_d, float *p1_d, float *p2_d, float *p3_d,
                              float *r0_d, float *r1_d, float *r2_d, float *r3_d,
                              float *mi_d, float *bi_d,
                              float *Sb_d, float *St_d, float *Sx_d, float *Sy_d, float *Sn_d,
                              float *Ttt_d, float *Ttx_d, float *Tty_d, float *Ttn_d,
                              float *Txx_d, float *Txy_d, float *Txn_d, float *Tyy_d,
                              float *Tyn_d, float *Tnn_d, parameters params)
{

  // components of Tmunu: Ttt, Ttx, Tty, Ttn, Txx, Txy, Txn, Tyy, Tyn, Tnn

  long int blockId = blockIdx.x + blockIdx.y * gridDim.x
                  + gridDim.x * gridDim.y * blockIdx.z;
  long int tid =     blockId * (blockDim.x * blockDim.y * blockDim.z)
                  + (threadIdx.z * (blockDim.x * blockDim.y))
                  + (threadIdx.y * blockDim.x) + threadIdx.x;

  int Ntot = params.NTOT;
  int nev = params.NEV;
  float sigma = params.SIGMA;
  float sigman = params.SIGMAN;
  float delta_tau = params.DELTA_TAU;
  float t0 = params.T0;
  int Nx = params.NX;
  int Ny = params.NY;
  int Nn = params.NN;
  float dt = params.DT;
  float dx = params.DX;
  float dy = params.DY;
  float dn = params.DN;
  
  if (tid < Ntot)
  {
    
    //reconstruct indices manually using
    // s = i + j * (Nx) + k * (Nx * Ny);
    int k = tid / (Nx * Ny);
    int j = ( tid - (k * Nx * Ny) ) / Nx;
    int i = tid - (k * Nx * Ny) - (j * Nx);

    //printf("tid = %d\n", tid);
    float tau = t0 + ((float)it) * dt;
    float tauInv = 1.0 / tau;
    float rr[4];

    float SigInv = 1.0/(2.0*sigma*sigma);
    float SignInv = 1.0/(2.0*sigman*sigman);
    float d_tauInv = 1.0/delta_tau;
    float nevInv = 1.0/(float)nev;
    float hbarcNevInv = nevInv/hbarc;

    float prefac = 1.0/(2.0 * (2.0*PI*sigma*sigma) * sqrt(2.0*PI*sigman*sigman));
    float prefactor = d_tauInv * prefac;
    float facN = prefactor * nevInv;
    float facHN = prefactor * hbarcNevInv;
    float hc2NevInv = nevInv/(hbarc*hbarc);

    //==========================================================================
    // calculate the source terms for rid ijk and write them in the output file

    float Sb = 0.0;
    float St = 0.0;
    float Sx = 0.0;
    float Sy = 0.0;
    float Sn = 0.0;

    float Ttt = 0.0;
    float Ttx = 0.0;
    float Tty = 0.0;
    float Ttn = 0.0;
    float Txx = 0.0;
    float Txy = 0.0;
    float Txn = 0.0;
    float Tyy = 0.0;
    float Tyn = 0.0;
    float Tnn = 0.0;

    rr[0] = tau;
    rr[1] = ((float)i - ((float)Nx - 1.0)/2.0) * dx;
    rr[2] = ((float)j - ((float)Ny - 1.0)/2.0) * dy;
    rr[3] = ((float)k - ((float)Nn - 1.0)/2.0) * dn;

    //==========================================================================
    // loop over all particles

    for (int m = 0; m < Npart; ++m)
    {
      if ( !isnan(r0_d[m]) )
      { //if the particle is outside the light cone, skip this particle
        float b_i = bi_d[m];
        float distt = fabs(rr[0]-r0_d[m]);
        if (distt < 4 * delta_tau)//if it's not far away in tau
        {
          float distn = fabs(rr[3]-r3_d[m]);
          if (distn < 3 * sigman)//if it's not far away in eta direction
          {
            float ddx = fabs(rr[1]-r1_d[m]);
            float ddy = fabs(rr[2]-r2_d[m]);
            float disttrs = ddx*ddx + ddy*ddy;
            float disttr = sqrt(disttrs);

            if (disttr < 3 * sigma)//if the particle is not far away in the transverse plane
            {
              // Smearing kernel
              float dist = -(disttrs * SigInv + distn*distn * SignInv);
              float numerator = exp(dist);
              float delta = distt * d_tauInv;
              float ch = cosh(delta);
              float kernel = 1.0/(ch * ch) * numerator; // [1]
              
              float mptau = mi_d[m]/p0_d[m];

              // Tmunu related
              float th = tanh((rr[0]-r0_d[m])*d_tauInv);
              float kernelT = (th + 1) * numerator;

              if ( !isnan(kernel) )
              { // if kernel is nan for some reasons, skip this particle
                // pi[m][4] is [GeV] by defination above
                Sb = Sb + kernel * mptau * b_i; // [1]
                St = St + kernel * mptau * p0_d[m]; // [GeV]
                Sx = Sx + kernel * mptau * p1_d[m]; // [GeV]
                Sy = Sy + kernel * mptau * p2_d[m]; // [GeV]
                Sn = Sn + kernel * mptau * p3_d[m] * tauInv; // [GeV/fm] caution, definition and tau here
              }

              if ( !isnan(kernelT) )
              {
                // components of Tmunu: Ttt, Ttx, Tty, Ttn, Txx, Txy, Txn, Tyy, Tyn, Tnn

                float ptauInv = 1/p0_d[m];
                Ttt = Ttt + kernelT * p0_d[m];
                Ttx = Ttx + kernelT * ptauInv * p0_d[m] * p1_d[m];
                Tty = Tty + kernelT * ptauInv * p0_d[m] * p2_d[m];
                Ttn = Ttn + kernelT * ptauInv * p0_d[m] * p3_d[m];
                Txx = Txx + kernelT * ptauInv * p1_d[m] * p1_d[m];
                Txy = Txy + kernelT * ptauInv * p1_d[m] * p2_d[m];
                Txn = Txn + kernelT * ptauInv * p1_d[m] * p3_d[m];
                Tyy = Tyy + kernelT * ptauInv * p2_d[m] * p2_d[m];
                Tyn = Tyn + kernelT * ptauInv * p2_d[m] * p3_d[m];
                Tnn = Tnn + kernelT * ptauInv * p3_d[m] * p3_d[m];
              }
            } //if (disttr < 3 * sigma)
          } //if (distn < 3 * sigman)
        } //if (distt < 4 * delta_tau)
      } //if ( !isnan(ri[m][0]) )
    } //for (int m = 0; m < N; ++m)

    //==========================================================================
    // Write the source terms to arrays
    Sb_d[tid] = facN  * Sb * tauInv; // [1/fm^4] = [1/fm^3] * [1] * [1/fm]
    St_d[tid] = facHN * St * tauInv; // [1/fm^5] = [1/(fm^4*GeV)] * [GeV] * [1/fm]
    Sx_d[tid] = facHN * Sx * tauInv; // [1/fm^5] = [1/(fm^4*GeV)] * [GeV] * [1/fm]
    Sy_d[tid] = facHN * Sy * tauInv; // [1/fm^5] = [1/(fm^4*GeV)] * [GeV] * [1/fm]
    Sn_d[tid] = facHN * Sn * tauInv; // [1/fm^6] = [1/(fm^4*GeV)] * [GeV/fm] * [1/fm]

    float hctfac = tauInv * hc2NevInv;

    Ttt_d[tid] = prefac * Ttt * hctfac;
    Ttx_d[tid] = prefac * Ttx * hctfac;
    Tty_d[tid] = prefac * Tty * hctfac;
    Ttn_d[tid] = prefac * Ttn * hctfac;
    Txx_d[tid] = prefac * Txx * hctfac;
    Txy_d[tid] = prefac * Txy * hctfac;
    Txn_d[tid] = prefac * Txn * hctfac;
    Tyy_d[tid] = prefac * Tyy * hctfac;
    Tyn_d[tid] = prefac * Tyn * hctfac;
    Tnn_d[tid] = prefac * Tnn * hctfac;
  } //if (tid < Ntot)
}
